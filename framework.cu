#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <cmath>

#include <vector>
#include <string_view>
#include <charconv>
#include <stdexcept>
#include <system_error>

#include <hip/hip_runtime.h>




// galaxy is stored as cartesian coordinates of its stars, each dimmension is in separate array
struct sGalaxy
{
    float* x;
    float* y;
    float* z;
};


class cuda_exception : public std::exception
{
    std::string_view msg;

public:
    explicit cuda_exception(std::string_view msg_) : msg(msg_) {}

    const char* what() const override
    {
        return msg.data();
    }
};



#include "kernel.cu"
#include "kernel_CPU.C"



// the size of the gallaxy can be arbitrary changed
#define N 2000 // [todo] use as command line parameter
// #define N 10



void generateGalaxies(sGalaxy A, sGalaxy B, int n) {
    for (int i = 0; i < n; i++) {
        // create star in A at random position first
        A.x[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
        A.y[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
        A.z[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
        // create star in B near star A
        // in small probability, create more displaced star
        if ((float)rand() / (float)RAND_MAX < 0.01f) {
            B.x[i] = A.x[i] + 10.0f * (float)rand() / (float)RAND_MAX;
            B.y[i] = A.y[i] + 10.0f * (float)rand() / (float)RAND_MAX;
            B.z[i] = A.z[i] + 10.0f * (float)rand() / (float)RAND_MAX;
        }
        else {
            B.x[i] = A.x[i] + 1.0f * (float)rand() / (float)RAND_MAX;
            B.y[i] = A.y[i] + 1.0f * (float)rand() / (float)RAND_MAX;
            B.z[i] = A.z[i] + 1.0f * (float)rand() / (float)RAND_MAX;
        }
    }
}


int main_exc(int argc, char** argv)
{
    // params
    int device = 0;

    bool run_cpu = true;
    bool run_gpu = true;

    bool seed_time = false;
    bool seed_number = false;
    unsigned int seed = 314159;

    // parse command line params
    std::vector<std::string_view> args(argv + 1, argv + argc);

    bool old_args_style = false;
    if (args.size() == 1) {
        // for compatibility with original framework
        std::string_view device_str = args[0];

        unsigned int device_try = 0;
        const char* device_str_last = device_str.data() + device_str.size();
        auto [ptr, err] = std::from_chars(device_str.data(), device_str_last, device_try);

        if (ptr == device_str_last && err == std::errc()) {
            device = device_try;
            old_args_style = true;
        }
    }

    if (!old_args_style) {
        auto args_it = args.begin();

        while (args_it != args.end()) {
            std::string_view arg = *args_it;
            args_it++;

            if (arg == "-c" || arg == "--cpu") {
                run_cpu = true;
            } else if (arg == "-g" || arg == "--gpu") {
                run_gpu = true;
            } else if (arg == "-nc" || arg == "--no-cpu") {
                run_cpu = false;
            } else if (arg == "-ng" || arg == "--no-gpu") {
                run_gpu = false;
            } else if (arg == "-t" || arg == "--seed-time") {
                seed_time = true;
            } else if (arg == "-n" || arg == "--seed-number") {
                seed_number = true;

                if (args_it == args.end()) {
                    throw std::invalid_argument("-n / --seed-number > unsigned int expected");
                }
                std::string_view seed_str = *args_it;
                args_it++;

                unsigned int seed_try = 0;
                const char* seed_str_last = seed_str.data() + seed_str.size();
                auto [ptr, err] = std::from_chars(seed_str.data(), seed_str_last, seed_try);

                if (ptr != seed_str_last || err != std::errc()) {
                    throw std::invalid_argument("-n / --seed-number > unsigned int expected");
                }

                seed = seed_try;
            } else if (arg == "-d" || arg == "--device") {
                if (args_it == args.end()) {
                    throw std::invalid_argument("-d / --device > int expected");
                }
                std::string_view device_str = *args_it;
                args_it++;

                unsigned int device_try = 0;
                const char* device_str_last = device_str.data() + device_str.size();
                auto [ptr, err] = std::from_chars(device_str.data(), device_str_last, device_try);

                if (ptr != device_str_last || err != std::errc()) {
                    throw std::invalid_argument("-d / --device > int expected");
                }

                device = device_try;
            } else {
                throw std::invalid_argument("unknown argument");
            }
        }
    }

    // setup seed
    if (seed_time) {
        srand(time(NULL));
    }

    if (seed_number) {
        srand(seed);
    }

    // setup device
    if (hipSetDevice(device) != hipSuccess) {
        throw cuda_exception("cannot set CUDA device");
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Using device %d: \"%s\"\n", device, deviceProp.name);

    // create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    sGalaxy A, B;
    A.x = A.y = A.z = B.x = B.y = B.z = NULL;
    sGalaxy dA, dB;
    dA.x = dA.y = dA.z = dB.x = dB.y = dB.z = NULL;

    // allocate and set host memory
    A.x = (float*)malloc(N*sizeof(A.x[0]));
    A.y = (float*)malloc(N*sizeof(A.y[0]));
    A.z = (float*)malloc(N*sizeof(A.z[0]));
    B.x = (float*)malloc(N*sizeof(B.x[0]));
    B.y = (float*)malloc(N*sizeof(B.y[0]));
    B.z = (float*)malloc(N*sizeof(B.z[0]));
    generateGalaxies(A, B, N);      
 
    // allocate and set device memory
    if (hipMalloc((void**)&dA.x, N*sizeof(dA.x[0])) != hipSuccess
    || hipMalloc((void**)&dA.y, N*sizeof(dA.y[0])) != hipSuccess
    || hipMalloc((void**)&dA.z, N*sizeof(dA.z[0])) != hipSuccess
    || hipMalloc((void**)&dB.x, N*sizeof(dB.x[0])) != hipSuccess
    || hipMalloc((void**)&dB.y, N*sizeof(dB.y[0])) != hipSuccess
    || hipMalloc((void**)&dB.z, N*sizeof(dB.z[0])) != hipSuccess) {
        fprintf(stderr, "Device memory allocation error!\n");
        goto cleanup;
    }
    hipMemcpy(dA.x, A.x, N*sizeof(dA.x[0]), hipMemcpyHostToDevice);
    hipMemcpy(dA.y, A.y, N*sizeof(dA.y[0]), hipMemcpyHostToDevice);
    hipMemcpy(dA.z, A.z, N*sizeof(dA.z[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.x, B.x, N*sizeof(dB.x[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.y, B.y, N*sizeof(dB.y[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.z, B.z, N*sizeof(dB.z[0]), hipMemcpyHostToDevice);
    
    float time;
    float diff_CPU, diff_GPU;

    if (run_cpu) {
        std::cout << "        [CPU solve]\n";

        hipEventRecord(start, 0);
        diff_CPU = solveCPU(A, B, N);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        std::cout << "    [result]\n";
        std::cout << "CPU performance: " << static_cast<float>(N) * static_cast<float>(N - 1) / (2.0f * time * 1e3f) << " megapairs/s\n";
        std::cout << "CPU result: " << diff_CPU << "\n";
        std::cout << "\n";
    }

    if (run_gpu) {
        std::cout << "        [GPU solve]\n";

        hipEventRecord(start, 0);

        // run it 10x for more accurately timing results
        for (int i = 0; i < 10; i++) {
            diff_GPU = solveGPU(dA, dB, N);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        std::cout << "    [result]\n";
        std::cout << "GPU performance: " << static_cast<float>(N) * static_cast<float>(N - 1) / (2.0f * time * 1e2f) << " megapairs/s\n";
        std::cout << "GPU result: " << diff_GPU << "\n";
        std::cout << "\n";
    }

    if (run_cpu && run_gpu) {
        std::cout << "        [result check]\n";
        std::cout << "CPU result: " << diff_CPU << "\n";
        std::cout << "GPU result: " << diff_GPU << "\n";

        // check GPU results
        if (std::abs((diff_CPU - diff_GPU) / ((diff_CPU + diff_GPU) / 2.0f)) < 0.01f) { // ???
            std::cout << "Test OK :)\n";
        } else {
            std::cout << "Test FAILED :(\n";
        }

        std::cout << "\n";
    }

cleanup:
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    if (dA.x) hipFree(dA.x);
    if (dA.y) hipFree(dA.y);
    if (dA.z) hipFree(dA.z);
    if (dB.x) hipFree(dB.x);
    if (dB.y) hipFree(dB.y);
    if (dB.z) hipFree(dB.z);
    if (A.x) free(A.x);
    if (A.y) free(A.y);
    if (A.z) free(A.z);
    if (B.x) free(B.x);
    if (B.y) free(B.y);
    if (B.z) free(B.z);

    return 0;
}


int main(int argc, char** argv)
{
    try {
        main_exc(argc, argv);
    } catch (const std::exception& e) {
        std::cerr << "ERROR: " << e.what() << "\n";
    } catch (...) {
        std::cerr << "ERROR: unknown exception\n";
    }
}