#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <cmath>

#include <cstdlib>
#include <cstring>
#include <cinttypes>
#include <vector>
#include <type_traits>
#include <stdexcept>
#include <system_error>

#include <hip/hip_runtime.h>




// galaxy is stored as cartesian coordinates of its stars, each dimmension is in separate array
struct sGalaxy
{
    float* x;
    float* y;
    float* z;
};


class cuda_exception : public std::exception
{
    const char* msg;

public:
    explicit cuda_exception(const char* msg_) : msg(msg_) {}

    const char* what() const noexcept override
    {
        return msg;
    }
};



#include "kernel.cu"
#include "kernel_CPU.C"



void generateGalaxies(sGalaxy A, sGalaxy B, int n) {
    for (int i = 0; i < n; i++) {
        // create star in A at random position first
        A.x[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
        A.y[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
        A.z[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
        // create star in B near star A
        // in small probability, create more displaced star
        if ((float)rand() / (float)RAND_MAX < 0.01f) {
            B.x[i] = A.x[i] + 10.0f * (float)rand() / (float)RAND_MAX;
            B.y[i] = A.y[i] + 10.0f * (float)rand() / (float)RAND_MAX;
            B.z[i] = A.z[i] + 10.0f * (float)rand() / (float)RAND_MAX;
        }
        else {
            B.x[i] = A.x[i] + 1.0f * (float)rand() / (float)RAND_MAX;
            B.y[i] = A.y[i] + 1.0f * (float)rand() / (float)RAND_MAX;
            B.z[i] = A.z[i] + 1.0f * (float)rand() / (float)RAND_MAX;
        }
    }
}


template<typename T>
std::enable_if_t<std::is_signed_v<T>, bool> str_to_num(const char* s, T& value)
{
    char* s_last;
    long long value_ll = std::strtoll(s, &s_last, 10); // not optimized for smaller types

    if (*s_last != '\0' || value_ll < static_cast<long long>(std::numeric_limits<T>::lowest()) || value_ll > static_cast<long long>(std::numeric_limits<T>::max())) {
        return false;
    }

    value = static_cast<T>(value_ll);
    return true;
}

template<typename T>
std::enable_if_t<std::is_unsigned_v<T>, bool> str_to_num(const char* s, T& value)
{
    char* s_last;
    unsigned long long value_ll = std::strtoull(s, &s_last, 10); // not optimized for smaller types

    if (*s_last != '\0' || value_ll > static_cast<unsigned long long>(std::numeric_limits<T>::max())) {
        return false;
    }

    value = static_cast<T>(value_ll);
    return true;
}


int main_exc(int argc, char** argv)
{
    std::cout << "using framework 14\n\n";

    // params
    int device = 0;

    bool run_cpu = true;
    bool run_gpu = true;

    bool seed_time = false;
    bool seed_number = false;
    unsigned int seed = 314159;

    size_t stars_count = 2000;

    // parse command line params
    std::vector<char*> args(argv + 1, argv + argc);

    bool old_args_style = false;
    if (args.size() == 1) {
        // for compatibility with original framework
        char* device_str = args[0];

        char* device_str_last;
        int device_try = std::strtol(device_str, &device_str_last, 10); // ignoring out of range

        if (*device_str_last == '\0') {
            device = device_try;
            old_args_style = true;
        }
    }

    if (!old_args_style) {
        auto args_it = args.begin();

        while (args_it != args.end()) {
            char* arg = *args_it;
            args_it++;

            if (std::strcmp(arg, "-c") == 0 || std::strcmp(arg, "--cpu") == 0) {
                run_cpu = true;
            } else if (std::strcmp(arg, "-g") == 0 || std::strcmp(arg, "--gpu") == 0) {
                run_gpu = true;
            } else if (std::strcmp(arg, "-nc") == 0 || std::strcmp(arg, "--no-cpu") == 0) {
                run_cpu = false;
            } else if (std::strcmp(arg, "-ng") == 0 || std::strcmp(arg, "--no-gpu") == 0) {
                run_gpu = false;
            } else if (std::strcmp(arg, "-t") == 0 || std::strcmp(arg, "--seed-time") == 0) {
                seed_time = true;
            } else if (std::strcmp(arg, "-n") == 0 || std::strcmp(arg, "--seed-number") == 0) {
                seed_number = true;

                if (args_it == args.end()) {
                    throw std::invalid_argument("-n / --seed-number > argument expected");
                }
                char* seed_str = *args_it;
                args_it++;

                if (!str_to_num(seed_str, seed)) {
                    throw std::invalid_argument("-n / --seed-number > unsigned int expected");
                }
            } else if (std::strcmp(arg, "-d") == 0 || std::strcmp(arg, "--device") == 0) {
                if (args_it == args.end()) {
                    throw std::invalid_argument("-d / --device > argument expected");
                }
                char* device_str = *args_it;
                args_it++;

                if (!str_to_num(device_str, device)) {
                    throw std::invalid_argument("-d / --device > int expected");
                }
            } else if (std::strcmp(arg, "-s") == 0 || std::strcmp(arg, "--stars") == 0) {
                if (args_it == args.end()) {
                    throw std::invalid_argument("-s / --stars > argument expected");
                }
                char* stars_count_str = *args_it;
                args_it++;

                if (!str_to_num(stars_count_str, stars_count)) {
                    throw std::invalid_argument("-s / --stars > size_t expected");
                }
            } else {
                throw std::invalid_argument("unknown argument");
            }
        }
    }

    // setup seed
    if (seed_number) {
        srand(seed);
    } else if (seed_time) {
        srand(time(NULL));
    }

    // setup device
    if (hipSetDevice(device) != hipSuccess) {
        throw cuda_exception("cannot set CUDA device");
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    std::cout << "        [using device]\n";
    std::cout << "index : " << device << "\n";
    std::cout << "name : " << deviceProp.name << "\n";
    std::cout << "\n";

    // create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    sGalaxy A, B;
    A.x = A.y = A.z = B.x = B.y = B.z = NULL;
    sGalaxy dA, dB;
    dA.x = dA.y = dA.z = dB.x = dB.y = dB.z = NULL;

    // allocate and set host memory
    A.x = (float*)malloc(stars_count*sizeof(A.x[0]));
    A.y = (float*)malloc(stars_count*sizeof(A.y[0]));
    A.z = (float*)malloc(stars_count*sizeof(A.z[0]));
    B.x = (float*)malloc(stars_count*sizeof(B.x[0]));
    B.y = (float*)malloc(stars_count*sizeof(B.y[0]));
    B.z = (float*)malloc(stars_count*sizeof(B.z[0]));
    generateGalaxies(A, B, stars_count);      
 
    // allocate and set device memory
    if (hipMalloc((void**)&dA.x, stars_count*sizeof(dA.x[0])) != hipSuccess
    || hipMalloc((void**)&dA.y, stars_count*sizeof(dA.y[0])) != hipSuccess
    || hipMalloc((void**)&dA.z, stars_count*sizeof(dA.z[0])) != hipSuccess
    || hipMalloc((void**)&dB.x, stars_count*sizeof(dB.x[0])) != hipSuccess
    || hipMalloc((void**)&dB.y, stars_count*sizeof(dB.y[0])) != hipSuccess
    || hipMalloc((void**)&dB.z, stars_count*sizeof(dB.z[0])) != hipSuccess) {
        fprintf(stderr, "Device memory allocation error!\n");
        goto cleanup;
    }
    hipMemcpy(dA.x, A.x, stars_count*sizeof(dA.x[0]), hipMemcpyHostToDevice);
    hipMemcpy(dA.y, A.y, stars_count*sizeof(dA.y[0]), hipMemcpyHostToDevice);
    hipMemcpy(dA.z, A.z, stars_count*sizeof(dA.z[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.x, B.x, stars_count*sizeof(dB.x[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.y, B.y, stars_count*sizeof(dB.y[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.z, B.z, stars_count*sizeof(dB.z[0]), hipMemcpyHostToDevice);
    
    float time;
    float diff_CPU, diff_GPU;

    if (run_cpu) {
        std::cout << "        [CPU solve]\n";

        hipEventRecord(start, 0);
        diff_CPU = solveCPU(A, B, stars_count);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        std::cout << "    [result]\n";
        std::cout << "CPU performance: " << static_cast<float>(stars_count) * static_cast<float>(stars_count - 1) / (2.0f * time * 1e3f) << " megapairs/s\n";
        std::cout << "CPU result: " << diff_CPU << "\n";
        std::cout << "\n";
    }

    if (run_gpu) {
        std::cout << "        [GPU solve]\n";

        hipEventRecord(start, 0);

        // run it 10x for more accurately timing results
        for (int i = 0; i < 10; i++) {
            diff_GPU = solveGPU(dA, dB, stars_count);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        std::cout << "    [result]\n";
        std::cout << "GPU performance: " << static_cast<float>(stars_count) * static_cast<float>(stars_count - 1) / (2.0f * time * 1e2f) << " megapairs/s\n";
        std::cout << "GPU result: " << diff_GPU << "\n";
        std::cout << "\n";
    }

    if (run_cpu && run_gpu) {
        std::cout << "        [result check]\n";
        std::cout << "CPU result: " << diff_CPU << "\n";
        std::cout << "GPU result: " << diff_GPU << "\n";

        // check GPU results
        if (std::abs((diff_CPU - diff_GPU) / ((diff_CPU + diff_GPU) / 2.0f)) < 0.01f) { // ???
            std::cout << "Test OK :)\n";
        } else {
            std::cout << "Test FAILED :(\n";
        }

        std::cout << "\n";
    }

cleanup:
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    if (dA.x) hipFree(dA.x);
    if (dA.y) hipFree(dA.y);
    if (dA.z) hipFree(dA.z);
    if (dB.x) hipFree(dB.x);
    if (dB.y) hipFree(dB.y);
    if (dB.z) hipFree(dB.z);
    if (A.x) free(A.x);
    if (A.y) free(A.y);
    if (A.z) free(A.z);
    if (B.x) free(B.x);
    if (B.y) free(B.y);
    if (B.z) free(B.z);

    return 0;
}


int main(int argc, char** argv)
{
    try {
        main_exc(argc, argv);
    } catch (const std::exception& e) {
        std::cerr << "ERROR: " << e.what() << "\n";
    } catch (...) {
        std::cerr << "ERROR: unknown exception\n";
    }
}
