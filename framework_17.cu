#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <cmath>

#include <vector>
#include <string_view>
#include <charconv>
#include <stdexcept>
#include <system_error>
#include <optional>

#include <hip/hip_runtime.h>




// galaxy is stored as cartesian coordinates of its stars, each dimmension is in separate array
struct sGalaxy
{
    float* x;
    float* y;
    float* z;
};


class cuda_exception : public std::exception
{
    std::string_view msg;

public:
    explicit cuda_exception(std::string_view msg_) : msg(msg_) {}

    const char* what() const noexcept override
    {
        return msg.data();
    }
};


template<typename T>
class cuda_deleter
{
    void operator()(T* obj) const noexcept
    {
        hipFree(obj);
    }
};



#include "kernel.cu"
#include "kernel_CPU.C"



void generateGalaxies(sGalaxy A, sGalaxy B, int n) {
    for (int i = 0; i < n; i++) {
        // create star in A at random position first
        A.x[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
        A.y[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
        A.z[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
        // create star in B near star A
        // in small probability, create more displaced star
        if ((float)rand() / (float)RAND_MAX < 0.01f) {
            B.x[i] = A.x[i] + 10.0f * (float)rand() / (float)RAND_MAX;
            B.y[i] = A.y[i] + 10.0f * (float)rand() / (float)RAND_MAX;
            B.z[i] = A.z[i] + 10.0f * (float)rand() / (float)RAND_MAX;
        }
        else {
            B.x[i] = A.x[i] + 1.0f * (float)rand() / (float)RAND_MAX;
            B.y[i] = A.y[i] + 1.0f * (float)rand() / (float)RAND_MAX;
            B.z[i] = A.z[i] + 1.0f * (float)rand() / (float)RAND_MAX;
        }
    }
}


template<typename T>
std::optional<T> str_to_num(std::string_view s)
{
    T value;
    const char* s_last = s.data() + s.size();
    auto [ptr, err] = std::from_chars(s.data(), s_last, value);

    if (ptr != s_last || err != std::errc()) {
        return std::nullopt;
    }

    return std::optional<T>(value);
}


int main_exc(int argc, char** argv)
{
    std::cout << "using framework 17\n\n";

    // params
    int device = 0;

    bool run_cpu = true;
    bool run_gpu = true;

    bool seed_time = false;
    bool seed_number = false;
    unsigned int seed = 314159;

    size_t stars_count = 2000;

    // parse command line params
    std::vector<std::string_view> args(argv + 1, argv + argc);

    bool old_args_style = false;
    if (args.size() == 1) {
        // for compatibility with original framework
        std::string_view device_str = args[0];

        unsigned int device_try = 0;
        const char* device_str_last = device_str.data() + device_str.size();
        auto [ptr, err] = std::from_chars(device_str.data(), device_str_last, device_try);

        if (ptr == device_str_last && err == std::errc()) {
            device = device_try;
            old_args_style = true;
        }
    }

    if (!old_args_style) {
        auto args_it = args.begin();

        while (args_it != args.end()) {
            std::string_view arg = *args_it;
            args_it++;

            if (arg == "-c" || arg == "--cpu") {
                run_cpu = true;
            } else if (arg == "-g" || arg == "--gpu") {
                run_gpu = true;
            } else if (arg == "-nc" || arg == "--no-cpu") {
                run_cpu = false;
            } else if (arg == "-ng" || arg == "--no-gpu") {
                run_gpu = false;
            } else if (arg == "-t" || arg == "--seed-time") {
                seed_time = true;
            } else if (arg == "-n" || arg == "--seed-number") {
                if (args_it == args.end()) {
                    throw std::invalid_argument("-n / --seed-number > argument expected");
                }

                seed_number = true;
                std::optional<unsigned int> seed_try = str_to_num<unsigned int>(*args_it);
                args_it++;

                if (!seed_try.has_value()) {
                    throw std::invalid_argument("-n / --seed-number > unsigned int expected");
                }

                seed = seed_try.value();
            } else if (arg == "-d" || arg == "--device") {
                if (args_it == args.end()) {
                    throw std::invalid_argument("-d / --device > argument expected");
                }
                
                std::optional<int> device_try = str_to_num<int>(*args_it);
                args_it++;

                if (!device_try.has_value()) {
                    throw std::invalid_argument("-d / --device > int expected");
                }

                device = device_try.value();
            } else if (arg == "-s" || arg == "--stars") {
                if (args_it == args.end()) {
                    throw std::invalid_argument("-s / --stars > argument expected");
                }
                
                std::optional<long long> stars_count_try = str_to_num<long long>(*args_it);
                args_it++;

                if (!stars_count_try.has_value()) {
                    throw std::invalid_argument("-s / --stars > long long expected");
                }

                stars_count = stars_count_try.value();
            } else {
                throw std::invalid_argument("unknown argument");
            }
        }
    }

    // setup seed
    if (seed_number) {
        srand(seed);
    } else if (seed_time) {
        srand(time(NULL));
    }

    // setup device
    if (hipSetDevice(device) != hipSuccess) {
        throw cuda_exception("cannot set CUDA device");
    }

    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, device);

    std::cout << "        [using device]\n";
    std::cout << "index : " << device << "\n";
    std::cout << "name : " << device_prop.name << "\n";
    std::cout << "\n";

    // create events for timing
    hipEvent_t start;
    hipEventCreate(&start);
    hipEvent_t stop;
    hipEventCreate(&stop);

    sGalaxy A, B;
    A.x = A.y = A.z = B.x = B.y = B.z = NULL;
    sGalaxy dA, dB;
    dA.x = dA.y = dA.z = dB.x = dB.y = dB.z = NULL;

    // allocate and set host memory
    A.x = (float*)malloc(stars_count*sizeof(A.x[0]));
    A.y = (float*)malloc(stars_count*sizeof(A.y[0]));
    A.z = (float*)malloc(stars_count*sizeof(A.z[0]));
    B.x = (float*)malloc(stars_count*sizeof(B.x[0]));
    B.y = (float*)malloc(stars_count*sizeof(B.y[0]));
    B.z = (float*)malloc(stars_count*sizeof(B.z[0]));
    generateGalaxies(A, B, stars_count);      
 
    // allocate and set device memory
    if (hipMalloc((void**)&dA.x, stars_count*sizeof(dA.x[0])) != hipSuccess
    || hipMalloc((void**)&dA.y, stars_count*sizeof(dA.y[0])) != hipSuccess
    || hipMalloc((void**)&dA.z, stars_count*sizeof(dA.z[0])) != hipSuccess
    || hipMalloc((void**)&dB.x, stars_count*sizeof(dB.x[0])) != hipSuccess
    || hipMalloc((void**)&dB.y, stars_count*sizeof(dB.y[0])) != hipSuccess
    || hipMalloc((void**)&dB.z, stars_count*sizeof(dB.z[0])) != hipSuccess) {
        fprintf(stderr, "Device memory allocation error!\n");
        goto cleanup;
    }
    hipMemcpy(dA.x, A.x, stars_count*sizeof(dA.x[0]), hipMemcpyHostToDevice);
    hipMemcpy(dA.y, A.y, stars_count*sizeof(dA.y[0]), hipMemcpyHostToDevice);
    hipMemcpy(dA.z, A.z, stars_count*sizeof(dA.z[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.x, B.x, stars_count*sizeof(dB.x[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.y, B.y, stars_count*sizeof(dB.y[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.z, B.z, stars_count*sizeof(dB.z[0]), hipMemcpyHostToDevice);
    
    float time;
    float diff_cpu;
    float diff_gpu;

    if (run_cpu) {
        std::cout << "        [CPU solve]\n";

        hipEventRecord(start, 0);
        diff_cpu = solveCPU(A, B, stars_count);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        std::cout << "    [result]\n";
        std::cout << "CPU performance: " << static_cast<float>(stars_count) * static_cast<float>(stars_count - 1) / (2.0f * time * 1e3f) << " megapairs/s\n";
        std::cout << "CPU result: " << diff_cpu << "\n";
        std::cout << "\n";
    }

    if (run_gpu) {
        std::cout << "        [GPU solve]\n";

        hipEventRecord(start, 0);

        // run it 10x for more accurately timing results
        for (int i = 0; i < 10; i++) {
            diff_gpu = solveGPU(dA, dB, stars_count);
        }

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        std::cout << "    [result]\n";
        std::cout << "GPU performance: " << static_cast<float>(stars_count) * static_cast<float>(stars_count - 1) / (2.0f * time * 1e2f) << " megapairs/s\n";
        std::cout << "GPU result: " << diff_gpu << "\n";
        std::cout << "\n";
    }

    if (run_cpu && run_gpu) {
        std::cout << "        [result check]\n";
        std::cout << "CPU result: " << diff_cpu << "\n";
        std::cout << "GPU result: " << diff_gpu << "\n";

        // check GPU results
        if (std::abs((diff_cpu - diff_gpu) / ((diff_cpu + diff_gpu) / 2.0f)) < 0.01f) { // ???
            std::cout << "Test OK :)\n";
        } else {
            std::cout << "Test FAILED :(\n";
        }

        std::cout << "\n";
    }

cleanup:
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    if (dA.x) hipFree(dA.x);
    if (dA.y) hipFree(dA.y);
    if (dA.z) hipFree(dA.z);
    if (dB.x) hipFree(dB.x);
    if (dB.y) hipFree(dB.y);
    if (dB.z) hipFree(dB.z);
    if (A.x) free(A.x);
    if (A.y) free(A.y);
    if (A.z) free(A.z);
    if (B.x) free(B.x);
    if (B.y) free(B.y);
    if (B.z) free(B.z);

    return 0;
}


int main(int argc, char** argv)
{
    try {
        main_exc(argc, argv);
    } catch (const std::exception& e) {
        std::cerr << "ERROR: " << e.what() << "\n";
    } catch (...) {
        std::cerr << "ERROR: unknown exception\n";
    }
}
