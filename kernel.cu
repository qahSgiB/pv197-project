#include "hip/hip_runtime.h"
//TODO kernel implementation

#include <iostream>



__device__ float total_diff = 0.0f;



// printf("[%d %d] [%d %d] k_total_diff = %f\n", g_x, g_y, b_x, b_y, k_total_diff);

__global__ void kernel_main_simple(sGalaxy galaxy_a, sGalaxy galaxy_b, int n) {
    int gs_x = gridDim.x;
    int gs_y = gridDim.y;
    int bs_x = blockDim.x;
    int bs_y = blockDim.y;
    int ts_x = gs_x * bs_x;
    int ts_y = gs_y * bs_y;

    int g_x = blockIdx.x;
    int g_y = blockIdx.y;
    int b_x = threadIdx.x;
    int b_y = threadIdx.y;
    int t_x = g_x * bs_x + b_x;
    int t_y = g_y * bs_y + b_y;

    int ks_x = (n - 1) / ts_x + 1; // [opt?] param
    int ks_y = (n - 1) / ts_y + 1;

    // float nf = n;

    float k_total_diff = 0.0f;

    for (int k_y = 0; k_y < ks_y; k_y++) {
        for (int k_x = 0; k_x < ks_x; k_x++) {
            int galaxy0_index = k_x * ts_x + t_x;
            int galaxy1_index = k_y * ts_y + t_y;

            if (galaxy0_index < n && galaxy1_index < n && galaxy0_index < galaxy1_index) {
                float galaxy0a_x = galaxy_a.x[galaxy0_index];
                float galaxy0a_y = galaxy_a.y[galaxy0_index];
                float galaxy0a_z = galaxy_a.z[galaxy0_index];

                float galaxy0b_x = galaxy_b.x[galaxy0_index];
                float galaxy0b_y = galaxy_b.y[galaxy0_index];
                float galaxy0b_z = galaxy_b.z[galaxy0_index];

                float galaxy1a_x = galaxy_a.x[galaxy1_index];
                float galaxy1a_y = galaxy_a.y[galaxy1_index];
                float galaxy1a_z = galaxy_a.z[galaxy1_index];

                float galaxy1b_x = galaxy_b.x[galaxy1_index];
                float galaxy1b_y = galaxy_b.y[galaxy1_index];
                float galaxy1b_z = galaxy_b.z[galaxy1_index];

                float dx = galaxy0a_x - galaxy1a_x;
                float dy = galaxy0a_y - galaxy1a_y;
                float dz = galaxy0a_z - galaxy1a_z;

                float diff_a2 = dx * dx + dy * dy + dz * dz;

                dx = galaxy0b_x - galaxy1b_x;
                dy = galaxy0b_y - galaxy1b_y;
                dz = galaxy0b_z - galaxy1b_z;

                float diff_b2 = dx * dx + dy * dy + dz * dz;

                float diff = sqrtf(diff_a2) - sqrtf(diff_b2);
                k_total_diff += diff * diff;
            }
        }
    }

    // k_total_diff /= nf * (nf - 1);
    atomicAdd(&total_diff, k_total_diff);
}

float solveGPU(sGalaxy A, sGalaxy B, int n) {
    //TODO kernel call and data manipulation

    size_t grid_dim_x = 256;
    size_t grid_dim_y = 256;

    size_t block_dim_x = 16;
    size_t block_dim_y = 16;

    size_t k_x = (n - 1) / (grid_dim_x * block_dim_x) + 1; // round up
    size_t k_y = (n - 1) / (grid_dim_y * block_dim_y) + 1;

    std::cout << "    [kernel params]\n";
    std::cout << "grid size : " << grid_dim_x << " x " << grid_dim_y << "\n";
    std::cout << "block size : " << block_dim_x << " x " << block_dim_y << "\n";
    std::cout << "k : " << k_x << " x " << k_y << "\n";
    std::cout << "\n";

    dim3 grid_size(grid_dim_x, grid_dim_y);
    dim3 block_size(block_dim_x, block_dim_y);

    float diff = 0.0f;
    hipMemcpyToSymbol(HIP_SYMBOL(total_diff), &diff, sizeof(float));

    kernel_main_simple<<<grid_size, block_size>>>(A, B, n);

    hipMemcpyFromSymbol(&diff, HIP_SYMBOL(total_diff), sizeof(float));

    std::cout << "teeeeeeeeeeeeeeeeeeeeest " << diff << "\n";

    float nf = n;
    diff = std::sqrt(diff / (nf * (nf - 1)));

    return diff;
}