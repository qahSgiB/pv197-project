#include "hip/hip_runtime.h"
//TODO kernel implementation

#include <iostream>



__device__ float total_diff = 0.0f;



// printf("[%d %d] [%d %d] k_total_diff = %f\n", g_x, g_y, b_x, b_y, k_total_diff);

__global__ void kernel_main_simple(sGalaxy galaxy_a, sGalaxy galaxy_b, int n)
{
    int gs_x = gridDim.x;
    int gs_y = gridDim.y;
    int bs_x = blockDim.x;
    int bs_y = blockDim.y;
    int ts_x = gs_x * bs_x;
    int ts_y = gs_y * bs_y;

    int g_x = blockIdx.x;
    int g_y = blockIdx.y;
    int b_x = threadIdx.x;
    int b_y = threadIdx.y;
    int t_x = g_x * bs_x + b_x;
    int t_y = g_y * bs_y + b_y;

    int ks_x = (n - 1) / ts_x + 1; // [opt?] param
    int ks_y = (n - 1) / ts_y + 1;

    float nf = n;

    float k_total_diff = 0.0f;

    int bottom_left_x = g_x * bs_x;
    int bottom_left_y = (g_y + 1) * bs_y - 1;

    int ky_start = bottom_left_y > bottom_left_x ? 0 : (bottom_left_x - bottom_left_y) / ts_y + 1;

    // if (b_x == 0 && b_y == 0) {
    //     printf("[%d %d] [%d %d] ky_start = %d\n", g_x, g_y, b_x, b_y, ky_start);
    // }

    for (int k_y = ky_start; k_y < ks_y; k_y++) {
        int kx_end = (bottom_left_y - bottom_left_x + k_y * ts_y - 1) / ts_x + 1;

        // if (b_x == 0 && b_y == 0) {
        //     printf("[%d %d] [%d %d] kx_end = %d\n", g_x, g_y, b_x, b_y, kx_end);
        // }

        for (int k_x = 0; k_x < kx_end; k_x++) {
            int galaxy0_index = k_x * ts_x + t_x;
            int galaxy1_index = k_y * ts_y + t_y;

            if (galaxy0_index < n && galaxy1_index < n && galaxy0_index < galaxy1_index) {
            // if (galaxy0_index < n && galaxy1_index < n) {
                float galaxy0a_x = galaxy_a.x[galaxy0_index];
                float galaxy0a_y = galaxy_a.y[galaxy0_index];
                float galaxy0a_z = galaxy_a.z[galaxy0_index];

                float galaxy0b_x = galaxy_b.x[galaxy0_index];
                float galaxy0b_y = galaxy_b.y[galaxy0_index];
                float galaxy0b_z = galaxy_b.z[galaxy0_index];

                float galaxy1a_x = galaxy_a.x[galaxy1_index];
                float galaxy1a_y = galaxy_a.y[galaxy1_index];
                float galaxy1a_z = galaxy_a.z[galaxy1_index];

                float galaxy1b_x = galaxy_b.x[galaxy1_index];
                float galaxy1b_y = galaxy_b.y[galaxy1_index];
                float galaxy1b_z = galaxy_b.z[galaxy1_index];

                float dx = galaxy0a_x - galaxy1a_x;
                float dy = galaxy0a_y - galaxy1a_y;
                float dz = galaxy0a_z - galaxy1a_z;

                float diff_a2 = dx * dx + dy * dy + dz * dz;

                dx = galaxy0b_x - galaxy1b_x;
                dy = galaxy0b_y - galaxy1b_y;
                dz = galaxy0b_z - galaxy1b_z;

                float diff_b2 = dx * dx + dy * dy + dz * dz;

                // float diff = __fsqrt_rz(diff_a2) - __fsqrt_rz(diff_b2);
                // k_total_diff += diff * diff;
                // k_total_diff += diff_a2 + diff_b2 - 2 * sqrtf(diff_a2 * diff_b2);
                k_total_diff -= 2 * sqrtf(diff_a2 * diff_b2);
                k_total_diff += diff_a2 + diff_b2;
            }
        }
    }

    k_total_diff /= nf;
    // printf("[%d %d] [%d %d] k_total_diff = %f\n", g_x, g_y, b_x, b_y, k_total_diff);
    atomicAdd(&total_diff, k_total_diff);
}



float solve_gpu_param(sGalaxy A, sGalaxy B, int n, size_t grid_dim_x, size_t grid_dim_y, size_t block_dim_x, size_t block_dim_y, bool enable_output)
{
    size_t total_dim_x = grid_dim_x * block_dim_x;
    size_t total_dim_y = grid_dim_y * block_dim_y;

    size_t k_x = (n - 1) / total_dim_x + 1; // round up
    size_t k_y = (n - 1) / total_dim_y + 1;

    if (enable_output) {
        std::cout << "    [kernel params]\n";
        std::cout << "grid size : " << grid_dim_x << " x " << grid_dim_y << "\n";
        std::cout << "block size : " << block_dim_x << " x " << block_dim_y << "\n";
        std::cout << "k : " << k_x << " x " << k_y << "\n";
        std::cout << "total size : " << total_dim_x * k_x << " x " << total_dim_y * k_y << "\n";
    }

    dim3 grid_size(grid_dim_x, grid_dim_y);
    dim3 block_size(block_dim_x, block_dim_y);

    float diff = 0.0f;
    hipMemcpyToSymbol(HIP_SYMBOL(total_diff), &diff, sizeof(float));

    kernel_main_simple<<<grid_size, block_size>>>(A, B, n);

    hipMemcpyFromSymbol(&diff, HIP_SYMBOL(total_diff), sizeof(float));

    if (enable_output) { std::cout << "teeeeeeeeeeeeeeeeeeeeest " << diff << "\n"; }

    float nf = n;
    // diff = std::sqrt(diff / (nf * (nf - 1)));
    // diff = std::sqrt(diff);
    diff = std::sqrt(diff / (nf - 1));

    if (enable_output) { std::cout << "\n"; }

    return diff;
}

float solveGPU(sGalaxy A, sGalaxy B, int n)
{
    //TODO kernel call and data manipulation
    return solve_gpu_param(A, B, n, 128, 64, 16, 16, false);
}
